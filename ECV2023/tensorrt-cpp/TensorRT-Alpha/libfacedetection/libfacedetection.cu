#include "hip/hip_runtime.h"
#include"libfacedetection.h"

void calFeatureMapSize(const cv::Size& size, float* featureMapSize)
{
    // channel dim
    int p1_c = 51;
    int p2_c = 34;
    int p3_c = 34;
    int p4_c = 51;

    int h0 = int(int((size.height + 1) / 2) / 2);
    int w0 = int(int((size.width + 1) / 2) / 2);

    // P1(downsample by 8)
    int p1_h = int(h0 / 2);
    int p1_w = int(w0 / 2);

    // P2(downsample by 16)
    int p2_h = int(p1_h / 2);
    int p2_w = int(p1_w / 2);

    // P3(downsample by 32)
    int p3_h = int(p2_h / 2);
    int p3_w = int(p2_w / 2);

    // P4(downsample by 64)
    int p4_h = int(p3_h / 2);
    int p4_w = int(p3_w / 2);

    // todo
    featureMapSize[0] = (float)p1_h;
    featureMapSize[1] = (float)p1_w;
    featureMapSize[2] = (float)p1_c;

    featureMapSize[3] = (float)p2_h;
    featureMapSize[4] = (float)p2_w;
    featureMapSize[5] = (float)p2_c;

    featureMapSize[6] = (float)p3_h;
    featureMapSize[7] = (float)p3_w;
    featureMapSize[8] = (float)p3_c;

    featureMapSize[9] = p4_h;
    featureMapSize[10] = p4_w;
    featureMapSize[11] = p4_c;
}

void calPriorBox(float* featureMapSize, const float* minSizes, const int* dim2, const cv::Size& size, float* priorBox)
{
    float steps[4] = { 8, 16, 32, 64 };
    /* std::vector<float> min_size;
     min_size.reserve(3);*/
    cv::Vec4f anchor; // cx cy s_ky s_kx

    int idx = 0;
    for (size_t k = 0; k < 4; k++) // todo
    {
        //min_size = minSizes[k];
        for (size_t i = 0; i < featureMapSize[k * 3 + 0]; i++)
        {
            for (size_t j = 0; j < featureMapSize[k * 3 + 1]; j++)
            {
                for (size_t m = 0; m < dim2[k]; m++)
                {
                    priorBox[idx++] = ((float)j + 0.5) * steps[k] / size.width; // cx
                    priorBox[idx++] = ((float)i + 0.5) * steps[k] / size.height; // cy
                    priorBox[idx++] = minSizes[k * 3 + m] / size.width; // s_kx
                    priorBox[idx++] = minSizes[k * 3 + m] / size.height; // s_ky
                }

            }

        }

    }

}

LibFaceDet::LibFaceDet(const utils::InitParameter& param) : m_param(param)
{
    // const params
    m_min_sizes_device = nullptr;    // 4 * 3
    m_feat_hw_host_device = nullptr; // 4 * 3
    m_prior_boxes_device = nullptr;  // 18984 * 4
    m_variances_device = nullptr;    // 2 * 1
    checkRuntime(hipMalloc(&m_min_sizes_device, 4 * 3 * sizeof(float)));
    checkRuntime(hipMalloc(&m_feat_hw_host_device, 4 * 3 * sizeof(float)));

    checkRuntime(hipMalloc(&m_variances_device, 2 * 1 * sizeof(float)));
    m_feat_hw_host = new float[4 * 3];     // 4 * 3

    // input
    m_input_src_device = nullptr;
    m_input_hwc_device = nullptr;
    checkRuntime(hipMalloc(&m_input_src_device, param.batch_size * 3 * param.src_h * param.src_w * sizeof(float)));
    checkRuntime(hipMalloc(&m_input_hwc_device, param.batch_size * 3 * param.src_h * param.src_w * sizeof(float)));
   
    // output
    m_output_loc_device = nullptr;
    m_output_conf_device = nullptr;
    m_output_iou_device = nullptr;
    m_output_objects_device = nullptr;
    m_output_objects_width = 17; // 17: xyxy(4) + score(1) + class(1) + keepflag(1) + landmarks(10)

    int output_objects_size = param.batch_size * (1 + param.topK * m_output_objects_width); // 1: count
    checkRuntime(hipMalloc(&m_output_objects_device, output_objects_size * sizeof(float)));
    m_output_objects_host = new float[output_objects_size];
    m_objectss.resize(param.batch_size);
}

LibFaceDet::~LibFaceDet()
{
    // const params
    checkRuntime(hipFree(m_min_sizes_device));
    checkRuntime(hipFree(m_feat_hw_host_device));
    checkRuntime(hipFree(m_prior_boxes_device));
    checkRuntime(hipFree(m_variances_device));
    delete[] m_feat_hw_host;
    delete[] m_prior_boxes_host;

    // input
    checkRuntime(hipFree(m_input_src_device));
    checkRuntime(hipFree(m_input_hwc_device));
   
    // output
    checkRuntime(hipFree(m_output_loc_device));
    checkRuntime(hipFree(m_output_conf_device));
    checkRuntime(hipFree(m_output_iou_device));
    checkRuntime(hipFree(m_output_objects_device));
    delete[] m_output_objects_host;
}

bool LibFaceDet::init(const std::vector<unsigned char>& trtFile)
{
    // 1. init engine & context
    if (trtFile.empty())
    {
        return false;
    }
    // runtime
    std::unique_ptr<nvinfer1::IRuntime> runtime =
        std::unique_ptr<nvinfer1::IRuntime>(nvinfer1::createInferRuntime(sample::gLogger.getTRTLogger()));
    if (runtime == nullptr)
    {
        return false;
    }
    // deserializeCudaEngine
    this->m_engine = std::unique_ptr<nvinfer1::ICudaEngine>(runtime->deserializeCudaEngine(trtFile.data(), trtFile.size()));

    if (this->m_engine == nullptr)
    {
        return false;
    }
    // context
    this->m_context = std::unique_ptr<nvinfer1::IExecutionContext>(this->m_engine->createExecutionContext());
    if (this->m_context == nullptr)
    {
        return false;
    }
    // binding dim
    this->m_context->setBindingDimensions(0, nvinfer1::Dims4(m_param.batch_size, 3, m_param.src_h, m_param.src_w));

    // 2. get output's dim
    auto get_area = [](const nvinfer1::Dims& dims) {
        int area = 1;
        for (int i = 1; i < dims.nbDims; i++)
        {
            if (dims.d[i] != 0)
            {
                area *= dims.d[i];
            }
        }
        return area;
    };

    m_output_loc_dims  = this->m_context->getBindingDimensions(1);
    m_output_conf_dims = this->m_context->getBindingDimensions(2);
    m_output_iou_dims  = this->m_context->getBindingDimensions(3);
    
    m_total_objects = m_output_loc_dims.d[1];  // 18984
   
    // 3. malloc
    checkRuntime(hipMalloc(&m_prior_boxes_device, m_total_objects * 4 * sizeof(float))); // const params
    m_prior_boxes_host = new float[m_total_objects * 4];  // 18984 * 4
    checkRuntime(hipMalloc(&m_output_loc_device, m_param.batch_size * m_total_objects * 14 * sizeof(float)));
    checkRuntime(hipMalloc(&m_output_conf_device,m_param.batch_size * m_total_objects * 2 * sizeof(float)));
    checkRuntime(hipMalloc(&m_output_iou_device, m_param.batch_size * m_total_objects * 1 * sizeof(float)));

    // 4. init const params
    // min sizes
    checkRuntime(hipMemcpy(m_min_sizes_device, m_min_sizes_host, sizeof(float) * 4 * 3, hipMemcpyHostToDevice));
    
    // feature map size
    calFeatureMapSize(cv::Size(m_param.src_w, m_param.src_h), m_feat_hw_host);
    checkRuntime(hipMemcpy(m_feat_hw_host_device, m_feat_hw_host, sizeof(float) * 4 * 3, hipMemcpyHostToDevice));
    
    // prior boxes
    calPriorBox(m_feat_hw_host, m_min_sizes_host, m_min_sizes_host_dim, cv::Size(m_param.src_w, m_param.src_h), m_prior_boxes_host);
    //cv::Mat bbox_tmp(m_total_objects, 4, CV_32FC1, m_prior_boxes_host);
    checkRuntime(hipMemcpy(m_prior_boxes_device, m_prior_boxes_host, sizeof(float) * m_total_objects * 4, hipMemcpyHostToDevice));

    // variances
    checkRuntime(hipMemcpy(m_variances_device, m_variances_host, sizeof(float) * 2, hipMemcpyHostToDevice));
    return true;
}

void LibFaceDet::check()
{
    // print inputs and outputs' dims
    int idx;
    nvinfer1::Dims dims;

    sample::gLogInfo << "the engine's info:" << std::endl;
    for (auto layer_name : m_param.input_output_names)
    {
        idx = this->m_engine->getBindingIndex(layer_name.c_str());
        dims = this->m_engine->getBindingDimensions(idx);
        sample::gLogInfo << "idx = " << idx << ", " << layer_name << ": ";
        for (int i = 0; i < dims.nbDims; i++)
        {
            sample::gLogInfo << dims.d[i] << ", ";
        }
        sample::gLogInfo << std::endl;
    }

    sample::gLogInfo << "the context's info:" << std::endl;
    for (auto layer_name : m_param.input_output_names)
    {
        idx = this->m_engine->getBindingIndex(layer_name.c_str());
        dims = this->m_context->getBindingDimensions(idx);
        sample::gLogInfo << "idx = " << idx << ", " << layer_name << ": ";
        for (int i = 0; i < dims.nbDims; i++)
        {
            sample::gLogInfo << dims.d[i] << ", ";
        }
        sample::gLogInfo << std::endl;
    }
}

void LibFaceDet::copy(const std::vector<cv::Mat>& imgsBatch)
{
    cv::Mat img_fp32 = cv::Mat::zeros(imgsBatch[0].size(), CV_32FC3); // todo 
    hipHostRegister(img_fp32.data, img_fp32.elemSize() * img_fp32.total(), hipHostRegisterPortable);

    // copy to device
    float* pi = m_input_src_device;
    //for (size_t i = 0; i < m_param.batch_size; i++)
    for (size_t i = 0; i < imgsBatch.size(); i++)
    {
        //std::vector<float> img_vec = std::vector<float>(imgsBatch[i].reshape(1, 1));
        imgsBatch[i].convertTo(img_fp32, CV_32FC3);
        checkRuntime(hipMemcpy(pi, img_fp32.data, sizeof(float) * 3 * m_param.src_h * m_param.src_w, hipMemcpyHostToDevice));
        /*imgsBatch[i].convertTo(imgsBatch[i], CV_32FC3);
        checkRuntime(hipMemcpy(pi, imgsBatch[i].data, sizeof(float) * 3 * m_param.src_h * m_param.src_w, hipMemcpyHostToDevice));*/
        pi += 3 * m_param.src_h * m_param.src_w;
    }

    hipHostUnregister(img_fp32.data);
}

void LibFaceDet::preprocess(const std::vector<cv::Mat>& imgsBatch)
{
    // 1. hwc2chw
    hwc2chwDevice(m_param.batch_size, m_input_src_device, m_param.src_w, m_param.src_h,
        m_input_hwc_device, m_param.src_w, m_param.src_h);
#if 0
    {

        float* phost = new float[3 * m_param.src_h * m_param.src_w];
        float* pdevice = m_input_hwc_device;
        for (size_t j = 0; j < imgsBatch.size(); j++)
        {
            checkRuntime(hipMemcpy(phost, pdevice + j * 3 * m_param.src_h * m_param.src_w,
                sizeof(float) * 3 * m_param.src_h * m_param.src_w, hipMemcpyDeviceToHost));

            cv::Mat tmp = imgsBatch[j].clone();

            cv::Mat b(m_param.src_h, m_param.src_w, CV_32FC1, phost);
            cv::Mat g(m_param.src_h, m_param.src_w, CV_32FC1, phost + 1 * m_param.src_h * m_param.src_w);
            cv::Mat r(m_param.src_h, m_param.src_w, CV_32FC1, phost + 2 * m_param.src_h * m_param.src_w);
            std::vector<cv::Mat> bgr{ b, g, r };
            cv::Mat ret;
            cv::merge(bgr, ret);
            ret.convertTo(ret, CV_8UC3);
            cv::imshow("ret", ret);
            cv::waitKey(1);

            cv::Mat img_ = imgsBatch[j].clone();
        }
        delete[] phost;

    }
#endif

}

bool LibFaceDet::infer()
{
    float* bindings[] = { m_input_hwc_device, m_output_loc_device, m_output_conf_device, m_output_iou_device};
    bool context = m_context->executeV2((void**)bindings);
    return context;
}

void LibFaceDet::postprocess(const std::vector<cv::Mat>& imgsBatch)
{
#if 0 // valid
    {
        float* phost_loc  = new float[m_total_objects * 14];
        float* phost_conf = new float[m_total_objects * 2];
        float* phost_iou  = new float[m_total_objects * 1];
        
        float* pdevice_loc  = m_output_loc_device;
        float* pdevice_conf = m_output_conf_device;
        float* pdevice_iou  = m_output_iou_device;
        for (size_t j = 0; j < imgsBatch.size(); j++)
        {
            checkRuntime(hipMemcpy(phost_loc,  pdevice_loc  + j * m_total_objects * 14, sizeof(float) * m_total_objects * 14, hipMemcpyDeviceToHost));
            checkRuntime(hipMemcpy(phost_conf, pdevice_conf + j * m_total_objects * 2,  sizeof(float) * m_total_objects * 2,  hipMemcpyDeviceToHost));
            checkRuntime(hipMemcpy(phost_iou,  pdevice_iou  + j * m_total_objects * 1,  sizeof(float) * m_total_objects * 1,  hipMemcpyDeviceToHost));
            //cv::Mat img_loc(m_total_objects, 14, CV_32FC1, phost_loc);

            //save to binary
            utils::saveBinaryFile(phost_loc,  m_total_objects * 14, "loc.bin");
            utils::saveBinaryFile(phost_conf, m_total_objects * 2,  "conf.bin");
            utils::saveBinaryFile(phost_iou,  m_total_objects * 1,  "iou.bin");

        }
        delete[] phost_loc;
        delete[] phost_conf;
        delete[] phost_iou;
    }
#endif // 0
    // decode
    decodeLibFaceDetDevice(
        m_min_sizes_device,
        m_feat_hw_host_device,
        m_prior_boxes_device,
        m_variances_device,

        m_param.src_w, m_param.src_h,
        m_param.conf_thresh, m_param.batch_size, m_total_objects,

        m_output_loc_device, 14,
        m_output_conf_device, 2,
        m_output_iou_device, 1,
        m_output_objects_device, m_output_objects_width, m_param.topK  // 17: xyxy(4) + score(1) + class(1) + keepflag(1) + landmarks(10)
    );
#if 0 // valid 
    {
        // loc
        float* p_loc_host = new float[m_total_objects * 14];
        float* p_loc_device = nullptr;
        p_loc_device = m_output_loc_device;
        for (size_t i = 0; i < imgsBatch.size(); i++)
        {
            checkRuntime(hipMemcpy(p_loc_host, p_loc_device + i * m_total_objects * 14,
                m_total_objects * 14 * sizeof(float), hipMemcpyDeviceToHost));
            cv::Mat img_loc_(m_total_objects, 14, CV_32FC1, p_loc_host); 
        }
        delete[] p_loc_host;

        // m_output_objects_device
        float* p_obj_host = new float[1 + m_output_objects_width * m_param.topK];   // 17: xyxy(4) + score(1) + class(1) + keepflag(1) + landmarks(10)
        float* p_obj_device = m_output_objects_device;
        for (size_t j = 0; j < imgsBatch.size(); j++)
        {
            checkRuntime(hipMemcpy(p_obj_host, p_obj_device + j * (1 + m_output_objects_width * m_param.topK), 
                sizeof(float) * (1 + m_output_objects_width * m_param.topK), hipMemcpyDeviceToHost));
            int num_candidates = p_obj_host[0];
            cv::Mat img_bbox1(m_param.topK, m_output_objects_width, CV_32FC1, p_obj_host + 1);
        }
        delete[] p_obj_host;
    }
#endif // 0

    // nms
    nmsDeviceV1(m_param, m_output_objects_device, m_output_objects_width, m_param.topK, m_output_objects_width * m_param.topK + 1);
#if 0 // valid
    {
        float* phost = new float[1 + m_output_objects_width * m_param.topK];   // 17: xyxy(4) + score(1) + class(1) + keepflag(1) + landmarks(10)
        float* pdevice = m_output_objects_device;
        for (size_t j = 0; j < imgsBatch.size(); j++)
        {
            checkRuntime(hipMemcpy(phost, pdevice + j * (1 + m_output_objects_width * m_param.topK),
                sizeof(float) * (1 + m_output_objects_width * m_param.topK), hipMemcpyDeviceToHost));
            int num_candidates = phost[0];
            cv::Mat img_bbox2(m_param.topK, m_output_objects_width, CV_32FC1, phost + 1);
        }
        delete[] phost;
    }
#endif // 0

    // copy result
    checkRuntime(hipMemcpy(m_output_objects_host, m_output_objects_device, 
        m_param.batch_size * sizeof(float)* (1 + m_output_objects_width * m_param.topK), hipMemcpyDeviceToHost));

    for (size_t bi = 0; bi < imgsBatch.size(); bi++)
    {
        int num_boxes = std::min((int)(m_output_objects_host + bi * (m_param.topK * m_output_objects_width + 1))[0], m_param.topK);
        for (size_t i = 0; i < num_boxes; i++)
        {
            float* ptr = m_output_objects_host + bi * (m_param.topK * m_output_objects_width + 1) + m_output_objects_width * i + 1;
            int keep_flag = ptr[6];
            if (keep_flag)
            {
                utils::Box bbox(ptr[0], ptr[1], ptr[2], ptr[3], ptr[4], (int)ptr[5], 5); // 5: 5 face landmark points
                bbox.land_marks.emplace_back(cv::Point2i(ptr[7], ptr[8]));
                bbox.land_marks.emplace_back(cv::Point2i(ptr[9], ptr[10]));
                bbox.land_marks.emplace_back(cv::Point2i(ptr[11], ptr[12]));
                bbox.land_marks.emplace_back(cv::Point2i(ptr[13], ptr[14]));
                bbox.land_marks.emplace_back(cv::Point2i(ptr[15], ptr[16]));

                m_objectss[bi].emplace_back(bbox); 
            }
        }

    }
}

std::vector<std::vector<utils::Box>> LibFaceDet::getObjectss() const
{
    return this->m_objectss;
}

void LibFaceDet::reset()
{
    checkRuntime(hipMemset(m_output_objects_device, 0, sizeof(float) * m_param.batch_size * (1 + m_output_objects_width * m_param.topK)));
    for (size_t bi = 0; bi < m_param.batch_size; bi++)
    {
        m_objectss[bi].clear();
    }
}

__global__
void decode_face_det_device_kernel(float* minSizes, float* feat_hw, float* priorBoxes, float* variances,
    int srcImgWidth, int srcImgHeight,
    float confThreshold, int batchSize, int srcHeight,
    float* srcLoc, int srcLocWidth, int srcLocArea,
    float* srcConf, int srcConfWidth, int srcConfArea,
    float* srcIou, int srcIouWidth, int srcIouArea,
    float* dst, int dstWidth, int topK, int dstArea)
{
    int dx = blockDim.x * blockIdx.x + threadIdx.x; // "src's rpw" dim
    int dy = blockDim.y * blockIdx.y + threadIdx.y; // "batch size" dim
    if (dx >= srcHeight || dy >= batchSize)
    {
        return;
    }
    // conf(softmax)
    float* pitem_conf = srcConf + dy * srcConfArea + dx * srcConfWidth;
    float* pitem_iou = srcIou + dy * srcIouArea + dx * srcIouWidth;
    if (pitem_iou[0] < 0)
    {
        pitem_iou[0] = 0;
    }
    if (pitem_iou[0] > 1)
    {
        pitem_iou[0] = 1;
    }
    float e0 = expf(pitem_conf[0]);
    float e1 = expf(pitem_conf[1]);
    float exp_sum = e0 + e1;
    //pitem_conf[0] = e0 / exp_sum;
    pitem_conf[1] = e1 / exp_sum;
    float score = sqrt(pitem_conf[1] * pitem_iou[0]);
    if (score <= confThreshold)
    {
        return;
    }

    int index = atomicAdd(dst + dy * dstArea, 1);

    //int index = atomicAdd(&(dst + dy * dstWidth)[0], 1);
    if (index >= topK) // dstHeight : topK
    {
        return;
    }

    // bbox
    float* pitem_loc = srcLoc + dy * srcLocArea + dx * srcLocWidth;
    pitem_loc[0] = priorBoxes[4 * dx] + pitem_loc[0] * variances[0] * priorBoxes[4 * dx + 2];
    pitem_loc[1] = priorBoxes[4 * dx + 1] + pitem_loc[1] * variances[0] * priorBoxes[4 * dx + 3];
    pitem_loc[2] = priorBoxes[4 * dx + 2] * expf(pitem_loc[2] * variances[1]);
    pitem_loc[3] = priorBoxes[4 * dx + 3] * expf(pitem_loc[3] * variances[1]);

    // xywh2xyxy
    pitem_loc[0] -= pitem_loc[2] / 2;
    pitem_loc[1] -= pitem_loc[3] / 2;
    pitem_loc[2] += pitem_loc[0];
    pitem_loc[3] += pitem_loc[1];

    pitem_loc[0] *= srcImgWidth;
    pitem_loc[1] *= srcImgHeight;
    pitem_loc[2] *= srcImgWidth;
    pitem_loc[3] *= srcImgHeight;

    // 5 landmarks
    pitem_loc[4] = (priorBoxes[4 * dx] + pitem_loc[4] * variances[0] * priorBoxes[4 * dx + 2]) * srcImgWidth;
    pitem_loc[6] = (priorBoxes[4 * dx] + pitem_loc[6] * variances[0] * priorBoxes[4 * dx + 2]) * srcImgWidth;
    pitem_loc[8] = (priorBoxes[4 * dx] + pitem_loc[8] * variances[0] * priorBoxes[4 * dx + 2]) * srcImgWidth;
    pitem_loc[10] = (priorBoxes[4 * dx] + pitem_loc[10] * variances[0] * priorBoxes[4 * dx + 2]) * srcImgWidth;
    pitem_loc[12] = (priorBoxes[4 * dx] + pitem_loc[12] * variances[0] * priorBoxes[4 * dx + 2]) * srcImgWidth;

    pitem_loc[5] = (priorBoxes[4 * dx + 1] + pitem_loc[5] * variances[0] * priorBoxes[4 * dx + 3]) * srcImgHeight;
    pitem_loc[7] = (priorBoxes[4 * dx + 1] + pitem_loc[7] * variances[0] * priorBoxes[4 * dx + 3]) * srcImgHeight;
    pitem_loc[9] = (priorBoxes[4 * dx + 1] + pitem_loc[9] * variances[0] * priorBoxes[4 * dx + 3]) * srcImgHeight;
    pitem_loc[11] = (priorBoxes[4 * dx + 1] + pitem_loc[11] * variances[0] * priorBoxes[4 * dx + 3]) * srcImgHeight;
    pitem_loc[13] = (priorBoxes[4 * dx + 1] + pitem_loc[13] * variances[0] * priorBoxes[4 * dx + 3]) * srcImgHeight;

    // get dst
    float* pitem_dst = dst + dy * dstArea + index * dstWidth + 1;  // note: not dx but index

    // method1
    //memcpy(pitem_dst, pitem_loc, srcLocWidth * sizeof(float)); // dstWidth:14 + 1
    //pitem_dst[14] = score;

    // method2
    /*pitem_dst[0] = pitem_loc[0];
    pitem_dst[1] = pitem_loc[1];
    pitem_dst[2] = pitem_loc[2];
    pitem_dst[3] = pitem_loc[3];
    pitem_dst[4] = pitem_loc[4];
    pitem_dst[5] = pitem_loc[5];
    pitem_dst[6] = pitem_loc[6];
    pitem_dst[7] = pitem_loc[7];
    pitem_dst[8] = pitem_loc[8];
    pitem_dst[9] = pitem_loc[9];
    pitem_dst[10] = pitem_loc[10];
    pitem_dst[11] = pitem_loc[11];
    pitem_dst[12] = pitem_loc[12];
    pitem_dst[13] = pitem_loc[13];
    pitem_dst[14] = score;*/

    // xyxy + landmarks + score -> // 17: xyxy(4) + score(1) + class(1) + keepflag(1) + landmarks(10)
    // xyxy
    pitem_dst[0] = pitem_loc[0];
    pitem_dst[1] = pitem_loc[1];
    pitem_dst[2] = pitem_loc[2];
    pitem_dst[3] = pitem_loc[3];
    // score
    pitem_dst[4] = score;
    // class
    pitem_dst[5] = 1; // 1:face, 0:no-face
    // feepflag
    pitem_dst[6] = 1; // 1:keep, 0: ignore
    // landmarks
    pitem_dst[7] = pitem_loc[4];
    pitem_dst[8] = pitem_loc[5];
    pitem_dst[9] = pitem_loc[6];
    pitem_dst[10] = pitem_loc[7];
    pitem_dst[11] = pitem_loc[8];
    pitem_dst[12] = pitem_loc[9];
    pitem_dst[13] = pitem_loc[10];
    pitem_dst[14] = pitem_loc[11];
    pitem_dst[15] = pitem_loc[12];
    pitem_dst[16] = pitem_loc[13];
}

void decodeLibFaceDetDevice(float* minSizes, float* feat_hw, float* priorBoxes, float* variances,
    int srcImgWidth, int srcImgHeight,
    float confThreshold, int batchSize, int srcHeight,
    float* srcLoc, int srcLocWidth,
    float* srcConf, int srcConfWidth,
    float* srcIou, int srcIouWidth,
    float* dst, int dstWidth, int dstHeight)
{
    dim3 block_size(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid_size((srcHeight + BLOCK_SIZE - 1) / BLOCK_SIZE,
        (batchSize + BLOCK_SIZE - 1) / BLOCK_SIZE);

    // todo
    int src_loc_area = srcHeight * srcLocWidth;   // 18984*14
    int src_conf_area = srcHeight * srcConfWidth; // 18984*2
    int src_iou_area = srcHeight * srcIouWidth;   // 18984*1
    int dst_area = dstHeight * dstWidth + 1;      // topK*17 + 1

    decode_face_det_device_kernel << < grid_size, block_size, 0, nullptr >> > (
        minSizes, feat_hw, priorBoxes, variances,
        srcImgWidth, srcImgHeight,
        confThreshold, batchSize, srcHeight,
        srcLoc, srcLocWidth, src_loc_area,
        srcConf, srcConfWidth, src_conf_area,
        srcIou, srcIouWidth, src_iou_area,
        dst, dstWidth, dstHeight, dst_area);
}
